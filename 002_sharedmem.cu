
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <random>

__global__ void add_vector(float *d_a, float *d_b, float *d_c, float p, float N)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
  {
    d_c[idx] = p * d_a[idx] + d_b[idx];
  }
}

int main()
{
  const int N = 10;

  float *a, *b, *c;
  hipMallocManaged(&a, N * sizeof(float));
  hipMallocManaged(&b, N * sizeof(float));
  hipMallocManaged(&c, N * sizeof(float));

  for (std::size_t i = 0; i < N; i++)
  {
    a[i] = (float)rand() / RAND_MAX;
    b[i] = (float)rand() / RAND_MAX;
  }

  const int block_size = 64;
  const int grid_size = (N + block_size - 1) / block_size;
  std::cout << "Grid size: " << grid_size << ". Block size: " << block_size << "." << std::endl;
  add_vector<<<grid_size, block_size>>>(a, b, c, 1.9f, N);

  // Wait for all previous operations before using values
  // We need this because we don't get the implicit synchronization of
  // cudaMemcpy like in the original example
  hipDeviceSynchronize();

  for (std::size_t i = 0; i < N; i++)
  {
    assert(c[i] == 1.9f * a[i] + b[i]);
  }

  hipFree(a);
  hipFree(b);
  hipFree(c);

  return 0;
}